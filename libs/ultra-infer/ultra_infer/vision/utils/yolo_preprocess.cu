
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// Part of the following code in this file refs to
// https://github.com/wang-xinyu/tensorrtx/blob/yolov5-v6.0/yolov5/preprocess.cu
//
// Copyright (c) 2022 tensorrtx
// Licensed under The MIT License
// \file preprocess.cu
// \brief
// \author Qi Liu, Xinyu Wang

#ifdef WITH_GPU
#include <opencv2/opencv.hpp>

#include "ultra_infer/vision/utils/cuda_utils.h"

namespace ultra_infer {
namespace vision {
namespace utils {

struct AffineMatrix {
  float value[6];
};

__global__ void
YoloPreprocessCudaKernel(uint8_t *src, int src_line_size, int src_width,
                         int src_height, float *dst, int dst_width,
                         int dst_height, uint8_t padding_color_b,
                         uint8_t padding_color_g, uint8_t padding_color_r,
                         AffineMatrix d2s, int edge) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= edge)
    return;

  float m_x1 = d2s.value[0];
  float m_y1 = d2s.value[1];
  float m_z1 = d2s.value[2];
  float m_x2 = d2s.value[3];
  float m_y2 = d2s.value[4];
  float m_z2 = d2s.value[5];

  int dx = position % dst_width;
  int dy = position / dst_width;
  float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
  float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
  float c0, c1, c2;

  if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
    // out of range
    c0 = padding_color_b;
    c1 = padding_color_g;
    c2 = padding_color_r;
  } else {
    int y_low = floorf(src_y);
    int x_low = floorf(src_x);
    int y_high = y_low + 1;
    int x_high = x_low + 1;

    uint8_t const_value[] = {padding_color_b, padding_color_g, padding_color_r};
    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    uint8_t *v1 = const_value;
    uint8_t *v2 = const_value;
    uint8_t *v3 = const_value;
    uint8_t *v4 = const_value;

    if (y_low >= 0) {
      if (x_low >= 0)
        v1 = src + y_low * src_line_size + x_low * 3;
      if (x_high < src_width)
        v2 = src + y_low * src_line_size + x_high * 3;
    }

    if (y_high < src_height) {
      if (x_low >= 0)
        v3 = src + y_high * src_line_size + x_low * 3;
      if (x_high < src_width)
        v4 = src + y_high * src_line_size + x_high * 3;
    }

    c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
    c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
    c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
  }

  // bgr to rgb
  float t = c2;
  c2 = c0;
  c0 = t;

  // normalization
  c0 = c0 / 255.0f;
  c1 = c1 / 255.0f;
  c2 = c2 / 255.0f;

  // rgbrgbrgb to rrrgggbbb
  int area = dst_width * dst_height;
  float *pdst_c0 = dst + dy * dst_width + dx;
  float *pdst_c1 = pdst_c0 + area;
  float *pdst_c2 = pdst_c1 + area;
  *pdst_c0 = c0;
  *pdst_c1 = c1;
  *pdst_c2 = c2;
}

void CudaYoloPreprocess(uint8_t *src, int src_width, int src_height, float *dst,
                        int dst_width, int dst_height,
                        const std::vector<float> padding_value,
                        cudaStream_t stream) {
  AffineMatrix s2d, d2s;
  float scale =
      std::min(dst_height / (float)src_height, dst_width / (float)src_width);

  s2d.value[0] = scale;
  s2d.value[1] = 0;
  s2d.value[2] = -scale * src_width * 0.5 + dst_width * 0.5;
  s2d.value[3] = 0;
  s2d.value[4] = scale;
  s2d.value[5] = -scale * src_height * 0.5 + dst_height * 0.5;

  cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
  cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
  cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);

  memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));

  int jobs = dst_height * dst_width;
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  YoloPreprocessCudaKernel<<<blocks, threads, 0, stream>>>(
      src, src_width * 3, src_width, src_height, dst, dst_width, dst_height,
      padding_value[0], padding_value[1], padding_value[2], d2s, jobs);
}

} // namespace utils
} // namespace vision
} // namespace ultra_infer
#endif
